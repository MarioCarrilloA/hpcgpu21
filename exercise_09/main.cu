#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <iomanip>
#include <math.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>

// Local headers
#include <kernels.h>

#define DEFAULT_NUM_ITERATIONS 1000
#define DEFAULT_NUM_PARTICLES  80000
#define DEFAULT_NUM_TO_SHOW    10
#define MAX_THREADS_PER_BLOCK 1024

using namespace std;

static const char help[] =
    "Usage: exercise09 [-k number] [-i number] [-p number] [-h]\n"
    "Description:\n"
    "  -i number:     Specifies how many times the kernel will be\n"
    "                 executed.\n"
    "  -p number:     Number of particles to be processed\n"
    "  -h             Prints this help message.\n";

void Print(p x) {
    for (int i = 0; i < DEFAULT_NUM_TO_SHOW; i++)
        cout << x.x[i] << endl;
}

void init(p xin, long npart) {
    for (int i = 0; i < npart; i++) {
        xin.x[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.y[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.z[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.m[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
    }
}

void execute_kernel(p xin, p xout, int npart, int niters) {
    p x_dev;
    p xin_dev;
    p xout_dev;
    float dt = 0.5f;
    float val = 0.5f;
    float execution_time = 0.0f;

    // Structures to measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Calculate blocks & threads
    int minimum_blocks;
    int minimum_threads;
    int extra_block;
    int total_blocks;

    // Minimum number of blocks/threads calculation
    if (npart < MAX_THREADS_PER_BLOCK) {
        minimum_blocks = 1;
        minimum_threads = npart;
    } else {
        minimum_blocks = npart / MAX_THREADS_PER_BLOCK;
        minimum_threads = MAX_THREADS_PER_BLOCK;
    }

    // Extra block calculation
    if (npart % MAX_THREADS_PER_BLOCK == 0 || npart < MAX_THREADS_PER_BLOCK)
        extra_block = 0;
    else
        extra_block = 1;

    total_blocks = minimum_blocks + extra_block;
    dim3 blocks(total_blocks, 1, 1);
    dim3 threads(minimum_threads, 1, 1);
    printf("Blocks:%d   Threads:%d\n", total_blocks, minimum_threads);
    printf( "Executing ...\n");

    // START measure time
    //hipEventRecord(start, 0);

    // Memory management
    checkCudaErrors(hipMalloc((void **)&xin_dev.x, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.y, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.z, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.m, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.x, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.y, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.z, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.m, sizeof(float) * npart));
    checkCudaErrors(hipMemcpy(xin_dev.x, xin.x, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.y, xin.y, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.z, xin.z, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.m, xin.m, sizeof(float) * npart, hipMemcpyHostToDevice));



    /// START measure time
    hipEventRecord(start, 0);

    // Kernel execution
    for (int i = 0; i < niters; i++) {
        kernel1<<<blocks, threads, sizeof(float) * 1024 * 12>>>(xin_dev, xout_dev, npart, dt, val);

        // Exchange pointers
        x_dev = xin_dev;
        xin_dev = xout_dev;
        xout_dev = x_dev;
    }

    // STOP measure time
    hipEventRecord(stop, 0);

    // Copy data from GPU to CPU to show results
    checkCudaErrors(hipMemcpy(xout.x, xout_dev.x, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.y, xout_dev.y, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.z, xout_dev.z, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.m, xout_dev.m, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(xin_dev.x));
    checkCudaErrors(hipFree(xin_dev.y));
    checkCudaErrors(hipFree(xin_dev.z));
    checkCudaErrors(hipFree(xin_dev.m));
    checkCudaErrors(hipFree(xout_dev.x));
    checkCudaErrors(hipFree(xout_dev.y));
    checkCudaErrors(hipFree(xout_dev.z));
    checkCudaErrors(hipFree(xout_dev.m));

    //Print(xout);

    // Calculate time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&execution_time, start, stop);
    printf("kernel execution: %f seconds\n", (execution_time / 1000.0f));
}


int exercise09(int npart, int niters) {
    p xin, xout;

    xin.x = (float*)malloc(sizeof(float) * npart);
    xin.y = (float*)malloc(sizeof(float) * npart);
    xin.z = (float*)malloc(sizeof(float) * npart);
    xin.m = (float*)malloc(sizeof(float) * npart);

    xout.x = (float*)malloc(sizeof(float) * npart);
    xout.y = (float*)malloc(sizeof(float) * npart);
    xout.z = (float*)malloc(sizeof(float) * npart);
    xout.m = (float*)malloc(sizeof(float) * npart);
    init(xin, npart);
    execute_kernel(xin, xout, npart, niters);
    free(xin.x);
    free(xin.y);
    free(xin.z);
    free(xin.m);
    free(xout.x);
    free(xout.y);
    free(xout.z);
    free(xout.m);

    return 0;
}


int main(int argc, char **argv) {
    int opt;
    int niters = DEFAULT_NUM_ITERATIONS;
    long int npart = DEFAULT_NUM_PARTICLES;

    while ((opt = getopt(argc, argv, "i:p:h")) != EOF) {
        switch (opt) {
            case 'i':
                niters = atoi(optarg);
                break;
            case 'p':
                npart = atoi(optarg);
                break;
            case 'h':
                cout << help << endl;
                return 0;
            case '?':
                cerr << "error: unknown option" << endl;
                cout << help << endl;
                return 1;
            default:
                cerr << help << endl;
                return 1;
        }
    }

    cout << "Particles: " << npart << endl;
    cout << "Iterations: " << niters << endl;
    exercise09(npart, niters);

    return 0;
}

