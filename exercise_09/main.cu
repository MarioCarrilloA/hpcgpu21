#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <iomanip>
#include <math.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>

// Local headers
#include <kernels.h>

#define DEFAULT_NUM_ITERATIONS 1000
#define DEFAULT_NUM_PARTICLES  80000
#define DEFAULT_NUM_TO_SHOW    10
#define MAX_THREADS_PER_BLOCK 1024

// Exercise 09 consists of 2 tasks. Therefore this number
// determines which one to execute
#define EXERCISE_TASK_NUM 2

using namespace std;

struct results {
    float E;
    float val;
    float dt;
};

static const char help[] =
    "Usage: exercise09 [-k number] [-i number] [-p number] [-h]\n"
    "Description:\n"
    "  -i number:     Specifies how many times the kernel will be\n"
    "                 executed.\n"
    "  -p number:     Number of particles to be processed\n"
    "  -h             Prints this help message.\n";

void Print(p x) {
    for (int i = 0; i < DEFAULT_NUM_TO_SHOW; i++)
        cout << x.x[i] << endl;
}

void init(p xin, long npart) {
    for (int i = 0; i < npart; i++) {
        xin.x[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.y[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.z[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.m[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
    }
}

float max_mass(p x, long npart) {
    float maxm = 0.0;
    for (int i = 0; i < npart; i++) {
        if (x.m[i] > maxm) {
            maxm = x.m[i];
        }
    }

    return maxm;
}

int execute_kernel(p xin, p xout, int npart, int niters) {
    p x_dev;
    p xin_dev;
    p xout_dev;
    float execution_time = 0.0f;

    // Structures to measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Calculate blocks & threads
    int minimum_blocks;
    int minimum_threads;
    int extra_block;
    int total_blocks;

    // Minimum number of blocks/threads calculation
    if (npart < MAX_THREADS_PER_BLOCK) {
        minimum_blocks = 1;
        minimum_threads = npart;
    } else {
        minimum_blocks = npart / MAX_THREADS_PER_BLOCK;
        minimum_threads = MAX_THREADS_PER_BLOCK;
    }

    // Extra block calculation
    if (npart % MAX_THREADS_PER_BLOCK == 0 || npart < MAX_THREADS_PER_BLOCK)
        extra_block = 0;
    else
        extra_block = 1;

    total_blocks = minimum_blocks + extra_block;
    dim3 blocks(total_blocks, 1, 1);
    dim3 threads(minimum_threads, 1, 1);
    printf("Blocks:%d   Threads:%d\n", total_blocks, minimum_threads);
    printf( "Executing ...\n");

    // GPU memory allocations/transfers
    checkCudaErrors(hipMalloc((void **)&xin_dev.x, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.y, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.z, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.m, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.x, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.y, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.z, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.m, sizeof(float) * npart));
    checkCudaErrors(hipMemcpy(xin_dev.x, xin.x, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.y, xin.y, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.z, xin.z, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.m, xin.m, sizeof(float) * npart, hipMemcpyHostToDevice));

    // #########################################################################
    // Stuff for all kernels
    // #########################################################################
    float *F;
    float *E;
    float *dt;
    float *val;

    // GPU vars
    float *F_dev;
    float *E_dev;
    float *dt_dev;
    float *val_dev;

    F = (float*)malloc(sizeof(float));
    E = (float*)malloc(sizeof(float));
    dt = (float*)malloc(sizeof(float));
    val = (float*)malloc(sizeof(float));
    *F = 0.0f;
    *E = 0.0f;
    *dt = 0.5f;
    *val = 0.5f;

    // Find max mass in particles
    float maxm = max_mass(xin, npart);
    float M = maxm * 1000.0f;

    // CPU results array
    results r[niters];
    // #########################################################################


    // The Exercise 09 is composed of 2 tasks, then the
    // kernels will be executed  according to task number.
    if (EXERCISE_TASK_NUM == 1) {
        // GPU memory allocations
        checkCudaErrors(hipMalloc((void **)&F_dev, sizeof(float)));
        checkCudaErrors(hipMalloc((void **)&E_dev, sizeof(float)));
        checkCudaErrors(hipMalloc((void **)&dt_dev, sizeof(float)));
        checkCudaErrors(hipMalloc((void **)&val_dev, sizeof(float)));
        checkCudaErrors(hipMemcpy(F_dev, F, sizeof(float), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(E_dev, E, sizeof(float), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dt_dev, dt, sizeof(float), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(val_dev, val, sizeof(float), hipMemcpyHostToDevice));

        // START measure time
        hipEventRecord(start, 0);

        // Kernel 2 - execution, calculate F
        kernel2<<<blocks, threads, 1024 * sizeof(float)>>>(xin_dev, F_dev, npart, M, dt_dev, val_dev);
        hipDeviceSynchronize();

        // Execute kernel sequence
        for (int i = 0; i < niters; i++) {
            // Kernel 1 - execution, exercise 04/07
            kernel1<<<blocks, threads, sizeof(float) * 1024 * 12>>>(xin_dev, xout_dev, npart, dt_dev, val_dev);
            hipDeviceSynchronize();

            // Kernel 2 - execution, calculate F
            kernel2<<<blocks, threads, 1024 * sizeof(float)>>>(xin_dev, F_dev, npart, M, dt_dev, val_dev);
            hipDeviceSynchronize();

            // Exchange pointers
            x_dev = xin_dev;
            xin_dev = xout_dev;
            xout_dev = x_dev;

            // Kernel 3 - execution  (NOW  / OLD), calculate E
            kernel3<<<blocks, threads, 1024 * sizeof(float)>>>(xout_dev, xin_dev, E_dev, npart);
            hipDeviceSynchronize();

            // Store values
            checkCudaErrors(hipMemcpy(E, E_dev, sizeof(float), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(dt, dt_dev, sizeof(float), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(val, val_dev, sizeof(float), hipMemcpyDeviceToHost));
            r[i].E = *E;
            r[i].dt = *dt;
            r[i].val = *val;
        }

        // STOP measure time
        hipEventRecord(stop, 0);

        // This just to hide a warning
        *dt = r[niters - 1].dt;


    } else if (EXERCISE_TASK_NUM == 2) {
        printf("Task 2\n");
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);

        if(prop.deviceOverlap) {
            printf("device porvides overlapping streams!!!\n");
        } else {
            printf("error: device provides NO overlapping streams\n");
            return 0;
        }

        hipStream_t stream1;
        hipStream_t stream2;
        hipStream_t stream3;
        hipStream_t stream4;
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);
        hipStreamCreate(&stream4);

        // GPU memory allocations
        checkCudaErrors(hipMalloc((void **)&F_dev, sizeof(float)));
        checkCudaErrors(hipMemcpy(F_dev, F, sizeof(float), hipMemcpyHostToDevice));

        // Asynchronous memory
        checkCudaErrors(hipHostMalloc((void **)&E_dev, sizeof(float)));
        checkCudaErrors(hipHostMalloc((void **)&dt_dev, sizeof(float)));
        checkCudaErrors(hipHostMalloc((void **)&val_dev, sizeof(float)));
        checkCudaErrors(hipMemcpyAsync(E_dev, E, sizeof(float), hipMemcpyHostToDevice, stream4));
        checkCudaErrors(hipMemcpyAsync(dt_dev, dt, sizeof(float), hipMemcpyHostToDevice, stream4));
        checkCudaErrors(hipMemcpyAsync(val_dev, val, sizeof(float), hipMemcpyHostToDevice, stream4));
        hipStreamSynchronize(stream4);

        // START measure time
        hipEventRecord(start, 0);

        // Kernel 2 - execution, calculate F
        kernel2<<<blocks, threads, 1024 * sizeof(float)>>>(xin_dev, F_dev, npart, M, dt_dev, val_dev);
        hipDeviceSynchronize();

        // Execute kernel sequence
        for (int i = 0; i < niters; i++) {
            // Kernel 1 - execution, exercise 04/07
            kernel1<<<blocks, threads, sizeof(float) * 1024 * 12, stream1>>>(xin_dev, xout_dev, npart, dt_dev, val_dev);
            hipDeviceSynchronize();

            // Kernel 2 - execution, calculate F
            kernel2<<<blocks, threads, 1024 * sizeof(float), stream2>>>(xin_dev, F_dev, npart, M, dt_dev, val_dev);
            hipDeviceSynchronize();

            // Exchange pointers
            x_dev = xin_dev;
            xin_dev = xout_dev;
            xout_dev = x_dev;

            // Kernel 3 - execution  (NOW  / OLD), calculate E
            kernel3<<<blocks, threads, 1024 * sizeof(float), stream3>>>(xout_dev, xin_dev, E_dev, npart);
            hipDeviceSynchronize();

            // Store values
            checkCudaErrors(hipMemcpyAsync(E, E_dev, sizeof(float), hipMemcpyDeviceToHost, stream4));
            checkCudaErrors(hipMemcpyAsync(dt, dt_dev, sizeof(float), hipMemcpyDeviceToHost, stream4));
            checkCudaErrors(hipMemcpyAsync(val, val_dev, sizeof(float), hipMemcpyDeviceToHost, stream4));
            hipStreamSynchronize(stream4);

            r[i].E = *E;
            r[i].dt = *dt;
            r[i].val = *val;
        }

        // STOP measure time
        hipEventRecord(stop, 0);
        hipHostFree(E);
        hipHostFree(dt);
        hipHostFree(val);

    } else {
        printf("error: Invalid task number\n");
    }


    // Copy data from GPU to CPU to show results
    checkCudaErrors(hipMemcpy(xout.x, xout_dev.x, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.y, xout_dev.y, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.z, xout_dev.z, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.m, xout_dev.m, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(xin_dev.x));
    checkCudaErrors(hipFree(xin_dev.y));
    checkCudaErrors(hipFree(xin_dev.z));
    checkCudaErrors(hipFree(xin_dev.m));
    checkCudaErrors(hipFree(xout_dev.x));
    checkCudaErrors(hipFree(xout_dev.y));
    checkCudaErrors(hipFree(xout_dev.z));
    checkCudaErrors(hipFree(xout_dev.m));

    //Print(xout);

    // Calculate time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&execution_time, start, stop);
    printf("kernel execution: %f seconds\n", (execution_time / 1000.0f));
    return 0;
}

int exercise09(int npart, int niters) {
    p xin, xout;

    xin.x = (float*)malloc(sizeof(float) * npart);
    xin.y = (float*)malloc(sizeof(float) * npart);
    xin.z = (float*)malloc(sizeof(float) * npart);
    xin.m = (float*)malloc(sizeof(float) * npart);

    xout.x = (float*)malloc(sizeof(float) * npart);
    xout.y = (float*)malloc(sizeof(float) * npart);
    xout.z = (float*)malloc(sizeof(float) * npart);
    xout.m = (float*)malloc(sizeof(float) * npart);
    init(xin, npart);
    execute_kernel(xin, xout, npart, niters);
    free(xin.x);
    free(xin.y);
    free(xin.z);
    free(xin.m);
    free(xout.x);
    free(xout.y);
    free(xout.z);
    free(xout.m);

    return 0;
}

int main(int argc, char **argv) {
    int opt;
    int niters = DEFAULT_NUM_ITERATIONS;
    long int npart = DEFAULT_NUM_PARTICLES;

    while ((opt = getopt(argc, argv, "i:p:h")) != EOF) {
        switch (opt) {
            case 'i':
                niters = atoi(optarg);
                break;
            case 'p':
                npart = atoi(optarg);
                break;
            case 'h':
                cout << help << endl;
                return 0;
            case '?':
                cerr << "error: unknown option" << endl;
                cout << help << endl;
                return 1;
            default:
                cerr << help << endl;
                return 1;
        }
    }

    cout << "Particles: " << npart << endl;
    cout << "Iterations: " << niters << endl;
    exercise09(npart, niters);

    return 0;
}

