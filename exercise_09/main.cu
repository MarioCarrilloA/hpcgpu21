#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <iomanip>
#include <math.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>

// Local headers
#include <kernels.h>

#define DEFAULT_NUM_ITERATIONS 1000
#define DEFAULT_NUM_PARTICLES  80000
#define DEFAULT_NUM_TO_SHOW    10
#define MAX_THREADS_PER_BLOCK 1024

using namespace std;

struct results {
    float E;
    float val;
    float dt;
};

static const char help[] =
    "Usage: exercise09 [-k number] [-i number] [-p number] [-h]\n"
    "Description:\n"
    "  -i number:     Specifies how many times the kernel will be\n"
    "                 executed.\n"
    "  -p number:     Number of particles to be processed\n"
    "  -h             Prints this help message.\n";

void Print(p x) {
    for (int i = 0; i < DEFAULT_NUM_TO_SHOW; i++)
        cout << x.x[i] << endl;
}

void init(p xin, long npart) {
    for (int i = 0; i < npart; i++) {
        xin.x[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.y[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.z[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
        xin.m[i] = (float(rand())/float((RAND_MAX)) * 10.0f) + 0.1f;
    }
}

float max_mass(p x, long npart) {
    float maxm = 0.0;
    for (int i = 0; i < npart; i++) {
        if (x.m[i] > maxm) {
            maxm = x.m[i];
        }
    }

    return maxm;
}

void execute_kernel(p xin, p xout, int npart, int niters) {
    p x_dev;
    p xin_dev;
    p xout_dev;
    float execution_time = 0.0f;

    // Structures to measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Calculate blocks & threads
    int minimum_blocks;
    int minimum_threads;
    int extra_block;
    int total_blocks;

    // Minimum number of blocks/threads calculation
    if (npart < MAX_THREADS_PER_BLOCK) {
        minimum_blocks = 1;
        minimum_threads = npart;
    } else {
        minimum_blocks = npart / MAX_THREADS_PER_BLOCK;
        minimum_threads = MAX_THREADS_PER_BLOCK;
    }

    // Extra block calculation
    if (npart % MAX_THREADS_PER_BLOCK == 0 || npart < MAX_THREADS_PER_BLOCK)
        extra_block = 0;
    else
        extra_block = 1;

    total_blocks = minimum_blocks + extra_block;
    dim3 blocks(total_blocks, 1, 1);
    dim3 threads(minimum_threads, 1, 1);
    printf("Blocks:%d   Threads:%d\n", total_blocks, minimum_threads);
    printf( "Executing ...\n");

    // GPU memory allocations/transfers
    checkCudaErrors(hipMalloc((void **)&xin_dev.x, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.y, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.z, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xin_dev.m, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.x, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.y, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.z, sizeof(float) * npart));
    checkCudaErrors(hipMalloc((void **)&xout_dev.m, sizeof(float) * npart));
    checkCudaErrors(hipMemcpy(xin_dev.x, xin.x, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.y, xin.y, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.z, xin.z, sizeof(float) * npart, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(xin_dev.m, xin.m, sizeof(float) * npart, hipMemcpyHostToDevice));

    // CPU results array
    results r[niters];

    // Stuff for kernel 1
    // #########################################################################
    float dt = 0.5f;
    float val = 0.5f;

    // Stuff for kernel 2
    // #########################################################################
    float *F;
    float *F_dev;

    // Find max mass in particles
    float maxm = max_mass(xin, npart);
    float M = maxm * 1000;

    F = (float*)malloc(sizeof(float));
    *F = 0.0f;
    checkCudaErrors(hipMalloc((void **)&F_dev, sizeof(float)));
    checkCudaErrors(hipMemcpy(F_dev, F, sizeof(float), hipMemcpyHostToDevice));


    // Stuff for kernel 3
    // #########################################################################
    float *E;
    float *E_dev;

    E = (float*)malloc(sizeof(float));
    *E = 0.0f;
    checkCudaErrors(hipMalloc((void **)&E_dev, sizeof(float)));
    checkCudaErrors(hipMemcpy(E_dev, E, sizeof(float), hipMemcpyHostToDevice));

    // START measure time
    hipEventRecord(start, 0);

    // Kernel 2 - execution, calculate F
    kernel2<<<blocks, threads, 1024 * sizeof(float)>>>(xin_dev, F_dev, npart, M);
    hipDeviceSynchronize();

    // Adjust val, dt
    checkCudaErrors(hipMemcpy(F, F_dev, sizeof(float), hipMemcpyDeviceToHost));
    val = *F;
    if ((val * dt) < 10.0f) {
        dt = dt * 0.1f;
    }

    // Kernel execution
    for (int i = 0; i < niters; i++) {
        // Kernel 1 - execution, exercise 04/07
        kernel1<<<blocks, threads, sizeof(float) * 1024 * 12>>>(xin_dev, xout_dev, npart, dt, val);
        hipDeviceSynchronize();

        // Kernel 2 - execution, calculate F
        kernel2<<<blocks, threads, 1024 * sizeof(float)>>>(xin_dev, F_dev, npart, M);
        hipDeviceSynchronize();

        // Adjust val, dt
        checkCudaErrors(hipMemcpy(F, F_dev, sizeof(float), hipMemcpyDeviceToHost));
        val = *F;
        if ((val * dt) < 10.0f) {
            dt = dt * 0.1f;
        }

        // Exchange pointers
        x_dev = xin_dev;
        xin_dev = xout_dev;
        xout_dev = x_dev;

        // Kernel 3 - execution  (NOW  / OLD), calculate E
        kernel3<<<blocks, threads, 1024 * sizeof(float)>>>(xout_dev, xin_dev, E_dev, npart);
        hipDeviceSynchronize();

        // Store values
        checkCudaErrors(hipMemcpy(E, E_dev, sizeof(float), hipMemcpyDeviceToHost));
        r[i].E = *E;
        r[i].dt = dt;
        r[i].val = val;
    }

    // STOP measure time
    hipEventRecord(stop, 0);

    // This just to hide a warning
    dt = r[npart - 1].dt;

    // Copy data from GPU to CPU to show results
    checkCudaErrors(hipMemcpy(xout.x, xout_dev.x, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.y, xout_dev.y, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.z, xout_dev.z, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(xout.m, xout_dev.m, sizeof(float) * npart, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(xin_dev.x));
    checkCudaErrors(hipFree(xin_dev.y));
    checkCudaErrors(hipFree(xin_dev.z));
    checkCudaErrors(hipFree(xin_dev.m));
    checkCudaErrors(hipFree(xout_dev.x));
    checkCudaErrors(hipFree(xout_dev.y));
    checkCudaErrors(hipFree(xout_dev.z));
    checkCudaErrors(hipFree(xout_dev.m));

    //Print(xout);

    // Calculate time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&execution_time, start, stop);
    printf("kernel execution: %f seconds\n", (execution_time / 1000.0f));
}

int exercise09(int npart, int niters) {
    p xin, xout;

    xin.x = (float*)malloc(sizeof(float) * npart);
    xin.y = (float*)malloc(sizeof(float) * npart);
    xin.z = (float*)malloc(sizeof(float) * npart);
    xin.m = (float*)malloc(sizeof(float) * npart);

    xout.x = (float*)malloc(sizeof(float) * npart);
    xout.y = (float*)malloc(sizeof(float) * npart);
    xout.z = (float*)malloc(sizeof(float) * npart);
    xout.m = (float*)malloc(sizeof(float) * npart);
    init(xin, npart);
    execute_kernel(xin, xout, npart, niters);
    free(xin.x);
    free(xin.y);
    free(xin.z);
    free(xin.m);
    free(xout.x);
    free(xout.y);
    free(xout.z);
    free(xout.m);

    return 0;
}

int main(int argc, char **argv) {
    int opt;
    int niters = DEFAULT_NUM_ITERATIONS;
    long int npart = DEFAULT_NUM_PARTICLES;

    while ((opt = getopt(argc, argv, "i:p:h")) != EOF) {
        switch (opt) {
            case 'i':
                niters = atoi(optarg);
                break;
            case 'p':
                npart = atoi(optarg);
                break;
            case 'h':
                cout << help << endl;
                return 0;
            case '?':
                cerr << "error: unknown option" << endl;
                cout << help << endl;
                return 1;
            default:
                cerr << help << endl;
                return 1;
        }
    }

    cout << "Particles: " << npart << endl;
    cout << "Iterations: " << niters << endl;
    exercise09(npart, niters);

    return 0;
}

