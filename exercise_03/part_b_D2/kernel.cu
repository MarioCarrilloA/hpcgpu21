#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "utils.h"

__global__ void kernel_draw_background(uchar3 *pos,int width, int height, int tick, p *earth) {
	int t = threadIdx.x;
    int g = blockIdx.x;
    int i = t + g * blockDim.x;
    int off = gridDim.x * blockDim.x;
    int N = width * height;

    while(i < N) {
        int currentX = i % width;
        int currentY = i / width;

        int Xs = (currentX - SUN_POS_X);
        int Ys = (currentY - SUN_POS_Y);
        //int Xe = (currentX - EARTH_POS_X);
        int Xe = (currentX - earth->x);
        //int Ye = (currentY - EARTH_POS_Y);
        int Ye = (currentY - earth->y);

        // Draw sun
        if (Xs * Xs + Ys * Ys <= SUN_RADIUS * SUN_RADIUS) {
            pos[i].x=255;
            pos[i].y=140;
            pos[i].z=0;

        // Draw earth
        } else if (Xe * Xe + Ye * Ye <= earth->radius * earth->radius) {
            pos[i].x=70;
            pos[i].y=130;
            pos[i].z=180;

        // Color background
        } else {
            pos[i].x=0;
            pos[i].y=0;
            pos[i].z=0;
        }

        i+=off;
    }
}


__global__ void kernel_draw_sun_particles(uchar3 *pos,int width, int height, p *particles) {
	int t = threadIdx.x;
    int g = blockIdx.x;
    int i = t + g * blockDim.x;
    int off = gridDim.x * blockDim.x;
    int N = width * height;

    while (i < MAX_PARTICLES) {
        int window_pos = (particles[i].y * height) - (width - particles[i].x);
        if (window_pos < N) {
            pos[window_pos].x=255;
            pos[window_pos].y=255;
            pos[window_pos].z=0;
        }
        i+=off;
    }

}


__global__ void kernel_update_particles_pos(uchar3 *pos,int width, int height, p *particles, p *earth) {
    int t = threadIdx.x;
    int g = blockIdx.x;
    int i = t + g * blockDim.x;
    int off = gridDim.x * blockDim.x;
    float time_step = 0.01;
    float ax = 0.0;
    float ay = 0.0;
    //double CONST = -0.015;
    double CONST = 0.015;

    earth->degree = earth->degree + 0.001;
    earth->x = ORBIT_POS_X + ORBIT_RADIUS * cos(earth->degree);
    earth->y = ORBIT_POS_Y + ORBIT_RADIUS * sin(earth->degree);

    if (earth->degree > 360.0)
        earth->degree = 0.0;

    while (i < MAX_PARTICLES) {
        double dx = abs(particles[i].x - earth->x);
        double dy = abs(particles[i].y - earth->y);
        double r = sqrt((dx * dx) + (dy * dy));

        // Aceleration
        ax = ((CONST * earth->mass) / (particles[i].mass * r)) * (particles[i].x / r);
        ay = ((CONST * earth->mass) / (particles[i].mass * r)) * (particles[i].y / r);

        particles[i].vx0 = particles[i].vx0 + (ax * time_step);
        particles[i].vy0 = particles[i].vy0 + (ay * time_step);
        particles[i].x = particles[i].x + particles[i].vx0;
        particles[i].y = particles[i].y + particles[i].vy0;

        // Validate if particle is out of our window width x height
       if (particles[i].y < 0 || particles[i].y > height || particles[i].x < 0 || particles[i].x > width) {
            particles[i].x = particles[i].default_x;
            particles[i].y = particles[i].default_y;
            particles[i].vx0 = particles[i].default_vx0;
            particles[i].vy0 = particles[i].default_vy0;
        }

        i+=off;
    }
}


void simulate(uchar3 *ptr, int tick, int w, int h, p *particles, p *earth)
{
	hipError_t err=hipSuccess;

	// set number of threads/blocks
	dim3 block(8,1,1);
	dim3 threads(1024,1,1);

    p *particles_dev;
    p *earth_dev;

	// Call kernel to draw sun, earth and color background
	//
    checkCudaErrors(hipMalloc((void **)&earth_dev, sizeof(p)));
    checkCudaErrors(hipMemcpy(earth_dev, earth, sizeof(p), hipMemcpyHostToDevice));



	kernel_draw_background<<<block, threads>>> (ptr, w, h, tick, earth_dev);
    checkCudaErrors(hipMalloc((void **)&particles_dev, sizeof(p) * MAX_PARTICLES));
    checkCudaErrors(hipMemcpy(particles_dev, particles, sizeof(p) * MAX_PARTICLES, hipMemcpyHostToDevice));

    // Call kernel to draw particles launched by the sun
    kernel_draw_sun_particles<<<block, threads>>> (ptr, w, h, particles_dev);

    // Call kernel to update values for the particles
    kernel_update_particles_pos<<<block, threads>>> (ptr, w, h, particles_dev, earth_dev);
    checkCudaErrors(hipMemcpy(earth, earth_dev, sizeof(p), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles, particles_dev, sizeof(p) * MAX_PARTICLES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(particles_dev));
	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//  stop in the GL-Loop to look at picture
	if(tick>=1) {
		//getchar();
	}
}
