#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
 
__global__ void kernel(uchar3 *pos,int width, int height) {
	// Implement the kernel here
}
void simulate(uchar3 *ptr, int tick, int w, int h)
{
	hipError_t err=hipSuccess;

	// set number of threads/blocks
	dim3 block(1,1,1);
	dim3 threads(1,1,1);

	// call your kernel
	kernel<<< block,threads>>> (ptr,w,h);
	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//  stop in the GL-Loop to look at picture
	if(tick>=1) {
		//getchar();
	}
}
