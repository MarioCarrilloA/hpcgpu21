#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
 
__global__ void kernel(uchar3 *pos,int width, int height, int tick) {
    // Implement the kernel here
    int t = threadIdx.x;
    int g = blockIdx.x;
    int i = t + g * blockDim.x;
    int off = gridDim.x * blockDim.x;
    int N = width * height;

    while(i < N) {
		int currentWid = i % width;
		int currentHei = i / width;
		if(currentWid * currentWid + currentHei * currentHei <= width * height){
			pos[i].x=0;
        	pos[i].y=0;
        	pos[i].z=255;
		}
		else{
			pos[i].x=255;
        	pos[i].y=0;
        	pos[i].z=0;
		}

		int checker = (currentWid / 32 + currentHei / 32) % 2;
		if(tick % 2 == 0){
			checker = 1 - checker;
		}

		if(checker == 0){
			pos[i].x=0;
        	pos[i].y=255;
        	pos[i].z=0;
		}
        i+=off;
    }
	
}
void simulate(uchar3 *ptr, int tick, int w, int h)
{
	hipError_t err=hipSuccess;

	// set number of threads/blocks
	dim3 block(1,1,1);
	dim3 threads(1,1,1);

	// call your kernel
	kernel<<< block,threads>>> (ptr,w,h, tick);
	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//  stop in the GL-Loop to look at picture
	if(tick>=1) {
		//getchar();
	}
}
