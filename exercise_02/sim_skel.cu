#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#define PI 3.1416
 
__global__ void kernel(uchar3 *pos,int width, int height, int tick) {
    // Implement the kernel here
    int t = threadIdx.x;
    int g = blockIdx.x;
    int i = t + g * blockDim.x;
    int off = gridDim.x * blockDim.x;
    int N = width * height;

    // Values for animated circle
    int xc = (tick % width) + 50;
    int yc = 100;
    int radius = 60;

    while (i < N) {
		int currentWid = i % width;
		int currentHei = i / width;
		if(currentWid * currentWid + currentHei * currentHei <= width * height){
			pos[i].x=0;
        	pos[i].y=0;
        	pos[i].z=255;
		}
		else {
			pos[i].x=255;
        	pos[i].y=0;
        	pos[i].z=0;
		}

		int checker = (currentWid / 32 + currentHei / 32) % 2;
		if (tick % 4 < 2){
			checker = 1 - checker;
		}

		if(checker == 0){
			pos[i].x=0;
        	pos[i].y=(tick+150) % 255;
        	pos[i].z=0;
		}

        // 199 and 601 will be the range of sine amplitude
        if (currentHei >= 199 && currentHei <= 601) {
            double degree = (currentWid * 360.0) / 800.0;
            double radians = degree * (PI/180.0);
            double amplitude = sin(radians);
            int pixely = ((amplitude + 1) * 400) / 2;
            // 199, 200 and 201 are bias values to do our line thicker
            if (pixely + 199 == currentHei
                || pixely + 200 == currentHei
                || pixely + 201 == currentHei ) {
                pos[i].x=255;
                pos[i].y=255;
                pos[i].z=0;
            }
        }

        int X = (currentWid - xc);
        int Y = (currentHei - yc);
        if (X * X + Y * Y <= radius * radius) {
            pos[i].x=255;
            pos[i].y=178;
            pos[i].z=0;
        }

        i+=off;
    }
}
void simulate(uchar3 *ptr, int tick, int w, int h)
{
	hipError_t err=hipSuccess;

	// set number of threads/blocks
	dim3 block(1,1,1);
	dim3 threads(1,1,1);

	// call your kernel
	kernel<<< block,threads>>> (ptr,w,h, tick);
	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//  stop in the GL-Loop to look at picture
	if(tick>=1) {
		//getchar();
	}
}
