#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
 
__global__ void kernel(uchar3 *pos,int width, int height) {
    // Implement the kernel here
    int t = threadIdx.x;
    int g = blockIdx.x;
    int i = t + g * blockDim.x;
    int off = gridDim.x * blockDim.x;
    int N = width * height;

    while(i < N) {
        pos[i].x=255;
        pos[i].y=0;
        pos[i].z=0;
        i+=off;
    }
}
void simulate(uchar3 *ptr, int tick, int w, int h)
{
	hipError_t err=hipSuccess;

	// set number of threads/blocks
	dim3 block(1,1,1);
	dim3 threads(1,1,1);

	// call your kernel
	kernel<<< block,threads>>> (ptr,w,h);
	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//  stop in the GL-Loop to look at picture
	if(tick>=1) {
		//getchar();
	}
}
